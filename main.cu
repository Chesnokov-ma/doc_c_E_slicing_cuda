
#include <iostream>
#include <fstream>
#include <vector>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <string>
#include <chrono>
#include <map>

#include <hip/hip_runtime.h>

using namespace std;

static void HandleError(hipError_t err, const char *file, int line)  // проверка на ошибку при операции с device-памятью на host
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))

#define N 40


__global__ void calc_Z_cuda(double* g_arr, double* E_arr, double* Z, double T, int count)       // расчет статсуммы
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < count)
    {
//        if (index < 1000)   printf("%d\t%lf\n", index, pow(2.7182, -29));

        atomicAdd(Z, g_arr[index] * pow(2.7182, (-1 * ((E_arr[index] + 0.17998) / T))));
    }
}

__global__ void calc_PE_cuda(double* g_arr, double* E_arr, double Z, double* PE, double T, int count)        // расчет массива вероятностей
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < count)
    {
//        printf("%d\t%lf\n", index, Z);
        PE[index] = (g_arr[index] * pow(2.7182, (-1 * ((E_arr[index] + 0.17998) / T)))) / Z;
    }
}

__global__ void calc_delta_E(double* E_arr, double* PE, double* delta_E, double* delta_E2, int count)        // расчет delta E и delta E2
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count)
    {
        atomicAdd(delta_E, PE[index] * E_arr[index]);
        atomicAdd(delta_E2, PE[index] * pow(E_arr[index], 2));
    }
}

int get_SP_cores(hipDeviceProp_t devProp)        // оптимальные размеры блока под используемую видеокарту
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

int main() {
    srand(time(NULL));

    string gem_file = "gem_true.txt";
    string ge_file = "ge.txt";
    string c_true_file = "c_true.txt";

    double difference = .0;              // обычная средняя разница
    double percent_difference = .0;      // средняя разница в процентах

    vector<double> C_true{}, C_current{};

    ifstream c_true(c_true_file);      // пересчитывать С при перевороте

    if (c_true.fail())
        throw invalid_argument(c_true_file + " not found");

    double tmp, tmp_c;

    while(c_true >> tmp >> tmp_c)
        C_true.push_back(tmp_c);

    c_true.close();
    FILE* diff = fopen("diff.txt", "w");

    // Загрузить dos, убрать столбец M -----------------------------------------------------------------------------------------------------------

    {
//
//    ifstream gem_input(gem_file);
//
//    if (gem_input.fail())
//        throw invalid_argument(gem_file + " not found");
//
//
//    map<double, int> GE;
//    GE.clear();
//
//    int gt; double et; int mt;
//    while(gem_input >> gt >> et >> mt)     // убрать столбец M
//    {
//        if (GE.count(et) == 0)        // если в словаре нет ключа Ei
//            GE[et] = gt;                    // создать с gi
//        else                                // если есть
//            GE[et] += gt;                   // добавить gi
//    }
//
//    gem_input.close();
//    ofstream ge_output(ge_file);
//
//    long long sum = 0;
//    for (pair<double, int> map_elem : GE)
//    {
//        ge_output << map_elem.second << "\t" << map_elem.first << endl;
//        sum += map_elem.second;
//    }
//
////    cout << pow(2, N) << "\t" << sum << endl;
//
//    ge_output.close();

    }

    // Загрузить ge -----------------------------------------------------------------------------------------------------------

    ifstream input(ge_file);

    double tmp0, tmp1;
    int rows = 0;
    while(input >> tmp0 >> tmp1)  rows++;       // читаю число строк из файла
    input.close();

    auto g = new double[rows];
    auto E = new double[rows];

    ifstream input1(ge_file);

    int count = 0;

    while(input1 >> tmp0 >> tmp1)
    {
        g[count] = tmp0;
        E[count] = tmp1;

        count++;
    }

    // Переменные CUDA-----------------------------------------------------------------------------------------

    int interval_num = rows;

    int block_dim = 512;
    int grid_dim_c = ((interval_num - 1) / block_dim) + 1;      // количество блоков для расчета теплоемкости (равно числу данных в файле)

    double Z = .0, *dev_Z;
    double *dev_dg_global, *dev_E_int_global, *dev_PE;

    auto PE = new double[interval_num - 1];                     // массив вероятностей
    double delta_E, delta_E2;
    double *dev_delta_E, *dev_delta_E2;

    HANDLE_ERROR(hipMalloc((void**)&dev_Z, sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_dg_global, (interval_num - 1) * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_E_int_global, (interval_num - 1) * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_PE, (interval_num - 1) * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_delta_E, sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_delta_E2, sizeof(double)));

    // Цикл-----------------------------------------------------------------------------------------------------

    // Каирская -> (, 36500)

    int skip_rows = 0;      // Сколько строк надо пропустить (нижн граница), все энергии в ge отсортированы по возрастанию map-ом и не повторяются
    int step = 25;          // Шаг по энергиям

    while( percent_difference < 0.1 )         // percent_difference < 0.1         skip_rows < rows           skip_rows < skip_rows + 1
    {
        if (skip_rows >= rows - 100)       // замедление на последних итерациях
            step = 1;

        int skip_rows_curr = skip_rows;

        for (int i = 0; i < skip_rows_curr; i++)        // срезание энергий
            g[rows - 1 - i] = 0;                          // начиная с макс
//            g[i] = 0;                                     // начиная с мин

        input1.close();

//        for (int i = 0; i < rows; i++)
//            cout << g[i] << "\t" << E[i] << endl;

        // Рассчитать теплоемкость для текущего ge файла ----------------------------------------------------------------------------------------------------------------------

        C_current.clear();

        FILE* f_c = fopen("c.txt", "w");

        HANDLE_ERROR(hipMemcpy(dev_dg_global, g, sizeof(double) * (interval_num - 1), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_E_int_global, E, sizeof(double) * (interval_num - 1), hipMemcpyHostToDevice));

        count = 0;

        double C = .0;

        for (double T = 0.00001; T < 0.12; T += 0.000001)
        {
            for (int i = 0; i < interval_num - 1; i++)  PE[i] = .0;

            HANDLE_ERROR(hipMemset(dev_Z, 0, sizeof(double)));
            HANDLE_ERROR(hipMemset(dev_delta_E, 0, sizeof(double)));
            HANDLE_ERROR(hipMemset(dev_delta_E2, 0, sizeof(double)));
            HANDLE_ERROR(hipMemcpy(dev_PE, PE, sizeof(double) * (interval_num - 1), hipMemcpyHostToDevice));

            calc_Z_cuda <<<grid_dim_c, block_dim>>>(dev_dg_global, dev_E_int_global, dev_Z, T, interval_num - 1);           // Z
            HANDLE_ERROR(hipMemcpy( &Z, dev_Z, sizeof(double), hipMemcpyDeviceToHost));

            calc_PE_cuda<<<grid_dim_c, block_dim>>>(dev_dg_global, dev_E_int_global, Z, dev_PE, T, interval_num - 1);       // PE
            HANDLE_ERROR(hipMemcpy( PE, dev_PE, sizeof(double) * (interval_num - 1), hipMemcpyDeviceToHost));

            if (Z != 0)
            {
                calc_delta_E<<<grid_dim_c, block_dim>>>(dev_E_int_global, dev_PE,dev_delta_E, dev_delta_E2, interval_num - 1);   // delta_E и delta_E2

                HANDLE_ERROR(hipMemcpy(&delta_E, dev_delta_E, sizeof(double), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipMemcpy(&delta_E2, dev_delta_E2, sizeof(double), hipMemcpyDeviceToHost));

                C = (delta_E2 - pow(delta_E, 2)) / pow(T, 2);
            }
            else
                C = 0;      // если CUDA отказывается правильно считать (Z == -inf)

//            cout << "T = " << T << "\tC = " << C << endl;
            fprintf(f_c, "%*.*lf\t%*.*lf\n", 5, 5, T, 5, 5, C);   // запись теплоемкости в файл

            C_current.push_back(round(C * 100000) / 100000);    // массив для сравнения с полными значениями

            T *= 1.05;
            count++;
        }

        fclose(f_c);

        // Найти разницу с эталоном ----------------------------------------------------------------------------------------------------------------------

        double sum = .0, sum_p = .0;
        int local_count = 0;

        int ignore_low_c = 10;      // игнорировать участки с ошибками в начале

        for (int i = ignore_low_c; i < count; i++)
        {
            if (C_true[i] != C_current[i])                                  // учитываются только различающиеся C
            {
                if (C_current[i] < C_true[i])
                {
                    sum += abs(C_true[i] - C_current[i]);
                    sum_p += abs(C_true[i] - C_current[i]) / C_true[i];

                    local_count++;
                }
            }

//            if (C_true[i] != C_current[i])
//                cout << i << " Diff: " <<  C_true[i] << "\t" << C_current[i] << "\t" << (C_true[i] - C_current[i]) / C_true[i] << endl;
//            else
//                cout << i << " " << C_true[i] << "\t" << C_current[i] << "\t" << (C_true[i] - C_current[i]) / C_true[i] << endl;
        }

//        cout << sum << "\t" << local_count << "\t" << sum / local_count << "\t" << sum_p << "\t" << sum_p / local_count << endl;

        if (local_count == 0)
        {
            difference = .0;
            percent_difference = .0;
        }
        else
        {
            difference = sum / local_count;                   // обычная средняя разница
            percent_difference = sum_p / local_count;         // средняя разница в процентах
        }

        cout << skip_rows << "\t" << difference << "\t" << percent_difference << endl;
        fprintf(diff, "%d\t%*.*lf\t%*.*lf\n", skip_rows, 5, 5, difference, 5, 5, percent_difference);   // запись в diff.txt


        skip_rows += step;
    }

    hipFree(dev_dg_global);
    hipFree(dev_E_int_global);
    hipFree(dev_PE);

    delete[] E;
    delete[] g;

    fclose(diff);

    return 0;
}
